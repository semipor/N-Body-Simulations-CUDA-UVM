#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <random>
#include "Constants.h"
#include <hip/hip_runtime.h>

void initializeBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass);
void runSimulation(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, char* image, float* hdImage);
//__global__ void interactBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass);
__global__ void clearF(float* Fx, float* Fy, float* Fz);
__global__ void interactn2(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* Fx, float* Fy, float* Fz);
__global__ void updateAll(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* Fx, float* Fy, float* Fz);
float magnitude(vec3 v);
void renderClear(char* image, float* hdImage);
__global__ void GPUrenderBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* hdImage);
float clamp(float x);
void writeRender(char* data, float* hdImage, int step);

int main()
{
	std::cout << SYSTEM_THICKNESS << "AU thick disk\n";;
	char *image;
	hipMallocManaged((void**)&image, sizeof(char)*WIDTH*HEIGHT*3);
	float *hdImage;
	hipMallocManaged((void**)&image, sizeof(fload)*WIDTH*HEIGHT*3);
	//struct body *bodies = new struct body[NUM_BODIES];
	
	float* xpos;
	float* ypos;
	float* zpos;
	float* xvel;
	float* yvel;
	float* zvel;
	float* mass;
	hipMallocManaged((void**)&xpos, sizeof(float)*NUM_BODIES);
	hipMallocManaged((void**)&ypos, sizeof(float)*NUM_BODIES);
	hipMallocManaged((void**)&zpos, sizeof(float)*NUM_BODIES);
	hipMallocManaged((void**)&xvel, sizeof(float)*NUM_BODIES);
	hipMallocManaged((void**)&yvel, sizeof(float)*NUM_BODIES);
	hipMallocManaged((void**)&zvel, sizeof(float)*NUM_BODIES);
	hipMallocManaged((void**)&mass, sizeof(float)*NUM_BODIES);

	initializeBodies(xpos,ypos,zpos,xvel,yvel,zvel,mass);
	runSimulation(xpos,ypos,zpos,xvel,yvel,zvel,mass, image, hdImage);
	std::cout << "\nwe made it\n";
	delete[] image;
	return 0;
}

void initializeBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass)
{
	using std::uniform_real_distribution;
	uniform_real_distribution<float> randAngle (0.0, 200.0*PI);
	uniform_real_distribution<float> randRadius (INNER_BOUND, SYSTEM_SIZE);
	uniform_real_distribution<float> randHeight (0.0, SYSTEM_THICKNESS);
	std::default_random_engine gen (0);
	float angle;
	float radius;
	float velocity;

	//STARS
	velocity = 0.67*sqrt((G*SOLAR_MASS)/(4*BINARY_SEPARATION*TO_METERS));
	//STAR 1
	xpos[0] = 0.0;///-BINARY_SEPARATION;
	ypos[0] = 0.0;
	zpos[0] = 0.0;
	xvel[0] = 0.0;
	yvel[0] = 0.0;//velocity;
	zvel[0] = 0.0;
	mass[0] = SOLAR_MASS;

	    ///STARTS AT NUMBER OF STARS///
	float totalExtraMass = 0.0;
	for (int i=1; i<NUM_BODIES; i++)
	{
		angle = randAngle(gen);
		radius = sqrt(SYSTEM_SIZE)*sqrt(randRadius(gen));
		velocity = pow(((G*(SOLAR_MASS+((radius-INNER_BOUND)/SYSTEM_SIZE)*EXTRA_MASS*SOLAR_MASS))
					  	  	  	  	  / (radius*TO_METERS)), 0.5);
		xpos[i] =  radius*cos(angle);
		ypos[i] =  radius*sin(angle);
		zpos[i] =  randHeight(gen)-SYSTEM_THICKNESS/2;
		xvel[i] =  velocity*sin(angle);
		yvel[i] = -velocity*cos(angle);
		zvel[i] =  0.0;
		mass[i] = (EXTRA_MASS*SOLAR_MASS)/NUM_BODIES;
		totalExtraMass += (EXTRA_MASS*SOLAR_MASS)/NUM_BODIES;
	}
	std::cout << "\nTotal Disk Mass: " << totalExtraMass;
	std::cout << "\nEach Particle weight: " << (EXTRA_MASS*SOLAR_MASS)/NUM_BODIES
			  << "\n______________________________\n";
}

void runSimulation(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, char* image, float* hdImage)
{
	float *Fx; float *Fy; float* Fz;
	hipMallocManaged(&Fx, NUM_BODIES*sizeof(float));
	hipMallocManaged(&Fy, NUM_BODIES*sizeof(float));
	hipMallocManaged(&Fz, NUM_BODIES*sizeof(float));
	int nBlocks=(NUM_BODIES+1024-1)/1024;
	long nsqrBlocks=(NUM_BODIES/1024)*(NUM_BODIES/1024);
	dim3 grid(nsqrBlocks,1024,1);
	//createFirstFrame
	renderClear(image, hdImage);
	GPUrenderBodies<<<nBlocks+1,1024>>>(xpos,ypos,zpos,xvel,yvel,zvel,mass,hdImage);
	writeRender(image, hdImage, 1);
	
	for (int step=1; step<STEP_COUNT; step++)
	{
		std::cout << "\nBeginning timestep: " << step;
		printf("\nStartK\n");
		hipMemcpy(d_xpos, xpos, NUM_BODIES*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_ypos, ypos, NUM_BODIES*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_zpos, zpos, NUM_BODIES*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_xvel, xvel, NUM_BODIES*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_yvel, yvel, NUM_BODIES*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_zvel, zvel, NUM_BODIES*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_mass, mass, NUM_BODIES*sizeof(float), hipMemcpyHostToDevice);
		printf("StartK1\n");	
		clearF<<<nBlocks+1,1024>>>(Fx,Fy,Fz);
		hipDeviceSynchronize();
		
		interactn2<<<grid,1024>>>(xpos,ypos,zpos,xvel,yvel,zvel,mass,Fx,Fy,Fz);
		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error!=hipSuccess)
		{
			printf("CUDA error:%s\n",hipGetErrorString(error));
		}
		updateAll<<<nBlocks+1,1024>>>(d_xpos,d_ypos,d_zpos,d_xvel,d_yvel,d_zvel,d_mass,Fx,Fy,Fz);
		printf("EndK\n");

		if (step%RENDER_INTERVAL==0)
		{
			std::cout << "\nWriting frame " << step;
			if (DEBUG_INFO)	{std::cout << "\nClearing Pixels..." << std::flush;}
			renderClear(image, hdImage);
			if (DEBUG_INFO) {std::cout << "\nRendering Particles..." << std::flush;}
			//renderBodies(pos, vel, hdImage);
			GPUrenderBodies<<<nBlocks+1,1024>>>(d_xpos,d_ypos,d_zpos,d_xvel,d_yvel,d_zvel,d_mass,d_hdImage);
			hipDeviceSynchronize();
			if (DEBUG_INFO) {std::cout << "\nWriting frame to file..." << std::flush;}
			writeRender(image, hdImage, step);
		}
		if (DEBUG_INFO) {std::cout << "\n-------Done------- timestep: "
			       << step << "\n" << std::flush;}
	}
}

__global__ void clearF(float* Fx, float* Fy, float* Fz)
{
	int id= blockIdx.x*blockDim.x+threadIdx.x;
	if(id<NUM_BODIES)
	{
		Fx[id]=0.0; Fy[id]=0.0; Fz[id]=0.0;
	}
}

__global__ void interactn2(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* Fx, float* Fy, float* Fz)
{
	long id = blockDim.x * blockIdx.x + threadIdx.x+ blockDim.y * blockIdx.y;
	long i=id%NUM_BODIES;
	long j=id/NUM_BODIES;
	if(i < NUM_BODIES && j< NUM_BODIES && i!=j)
	{	
		float v1 = (xpos[i]-xpos[j])*TO_METERS;
		float v2 = (ypos[i]-ypos[j])*TO_METERS;
		float v3 = (zpos[i]-zpos[j])*TO_METERS;
		float dist = sqrt(v1*v1+v2*v2+v3*v3);
		float F = TIME_STEP*(G*mass[i]*mass[j]) / ((dist*dist + SOFTENING*SOFTENING) * dist);
		atomicAdd(&Fx[i],F*v1/mass[i]);
		atomicAdd(&Fy[i],F*v2/mass[i]);
		atomicAdd(&Fz[i],F*v3/mass[i]);
	}
}

__global__ void updateAll(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* Fx, float* Fy, float* Fz)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<NUM_BODIES)
	{
		xvel[i] -= Fx[i];
		yvel[i] -= Fy[i];
		zvel[i] -= Fz[i];
		xpos[i] += TIME_STEP*xvel[i]/TO_METERS;
		ypos[i] += TIME_STEP*yvel[i]/TO_METERS;
		zpos[i] += TIME_STEP*zvel[i]/TO_METERS;
	}
}

float magnitude(vec3 v)
{
	return sqrt(v.x*v.x+v.y*v.y+v.z*v.z);
}

void renderClear(char* image, float* hdImage)
{
	for (int i=0; i<WIDTH*HEIGHT*3; i++)
	{
		image[i] = 0; //char(image[i]/1.2);
		hdImage[i] = 0.0;
	}
}

__global__ void GPUrenderBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* hdImage)
{
	/// ORTHOGONAL PROJECTION
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	float velocityMax = MAX_VEL_COLOR; //35000
	float velocityMin = sqrt(0.8*(G*(SOLAR_MASS+EXTRA_MASS*SOLAR_MASS))/
				(SYSTEM_SIZE*TO_METERS)); //MIN_VEL_COLOR;
	if(i<NUM_BODIES)
	{
		float vxsqr=xvel[i]*xvel[i];
		float vysqr=yvel[i]*yvel[i];
		float vzsqr=zvel[i]*zvel[i];
		float vMag = sqrt(vxsqr+vysqr+vzsqr);
		int x = (WIDTH/2.0)*(1.0+xpos[i]/(SYSTEM_SIZE*RENDER_SCALE));
		int y = (HEIGHT/2.0)*(1.0+ypos[i]/(SYSTEM_SIZE*RENDER_SCALE));

		if (x>DOT_SIZE && x<WIDTH-DOT_SIZE && y>DOT_SIZE && y<HEIGHT-DOT_SIZE)
		{
			float vPortion = sqrt((vMag-velocityMin) / velocityMax);
			float xPixel = (WIDTH/2.0)*(1.0+xpos[i]/(SYSTEM_SIZE*RENDER_SCALE));
			float yPixel = (HEIGHT/2.0)*(1.0+ypos[i]/(SYSTEM_SIZE*RENDER_SCALE));
			float xP = floor(xPixel);
			float yP = floor(yPixel);
			color c;
			c.r = max(min(4*(vPortion-0.333),1.0),0.0);
                        c.g = max(min(min(4*vPortion,4.0*(1.0-vPortion)),1.0),0.0);
                        c.b = max(min(4*(0.5-vPortion),1.0),0.0);
			for (int a=-DOT_SIZE/2; a<DOT_SIZE/2; a++)
			{
				for (int b=-DOT_SIZE/2; b<DOT_SIZE/2; b++)
				{
					float cFactor = PARTICLE_BRIGHTNESS /(pow(exp(pow(PARTICLE_SHARPNESS*(xP+a-xPixel),2.0)) + exp(pow(PARTICLE_SHARPNESS*(yP+b-yPixel),2.0)),/*1.25*/0.75)+1.0);
					//colorAt(int(xP+a),int(yP+b),c, cFactor, hdImage);
					int pix = 3*(xP+a+WIDTH*(yP+b));
					hdImage[pix+0] += c.r*cFactor;
					hdImage[pix+1] += c.g*cFactor;
					hdImage[pix+2] += c.b*cFactor;
				}
			}
		}
	}
}

float clamp(float x)
{
	return max(min(x,1.0),0.0);
}

void writeRender(char* data, float* hdImage, int step)
{
	
	for (int i=0; i<WIDTH*HEIGHT*3; i++)
	{
		data[i] = int(255.0*clamp(hdImage[i]));
	}

	int frame = step/RENDER_INTERVAL + 1;//RENDER_INTERVAL;
	std::string name = "images/Step"; 
	int i = 0;
	if (frame == 1000) i++; // Evil hack to avoid extra 0 at 1000
	for (i; i<4-floor(log(frame)/log(10)); i++)
	{
		name.append("0");
	}
	name.append(std::to_string(frame));
	name.append(".ppm");

	std::ofstream file (name, std::ofstream::binary);

	if (file.is_open())
	{
//		size = file.tellg();
		file << "P6\n" << WIDTH << " " << HEIGHT << "\n" << "255\n";
		file.write(data, WIDTH*HEIGHT*3);
		file.close();
	}

}

