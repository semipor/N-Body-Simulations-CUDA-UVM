#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <random>
#include "Constants.h"
#include <hip/hip_runtime.h>

void initializeBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass);
void runSimulation(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, char* image, float* hdImage);
__global__ void interactBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass);
float magnitude(vec3 v);
void renderClear(char* image, float* hdImage);
__global__ void GPUrenderBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* hdImage);
float clamp(float x);
void writeRender(char* data, float* hdImage, int step);

int main()
{
	std::cout << SYSTEM_THICKNESS << "AU thick disk\n";
	char *image;
	hipMallocManaged(&image, sizeof(char)*WIDTH*HEIGHT*3);
	float *hdImage;
	hipMallocManaged(&hdImage, sizeof(float)*WIDTH*HEIGHT*3);
	//struct body *bodies = new struct body[NUM_BODIES];
	
	float* xpos;
	float* ypos;
	float* zpos;
	float* xvel;
	float* yvel;
	float* zvel;
	float* mass;
	hipMallocManaged(&xpos, sizeof(float)*NUM_BODIES);
	hipMallocManaged(&ypos, sizeof(float)*NUM_BODIES);
	hipMallocManaged(&zpos, sizeof(float)*NUM_BODIES);
	hipMallocManaged(&xvel, sizeof(float)*NUM_BODIES);
	hipMallocManaged(&yvel, sizeof(float)*NUM_BODIES);
	hipMallocManaged(&zvel, sizeof(float)*NUM_BODIES);
	hipMallocManaged(&mass, sizeof(float)*NUM_BODIES);
	printf("MallocManaged successed");
	initializeBodies(xpos,ypos,zpos,xvel,yvel,zvel,mass);
	runSimulation(xpos,ypos,zpos,xvel,yvel,zvel,mass, image, hdImage);
	std::cout << "\nwe made it\n";
	delete[] image;
	return 0;
}

void initializeBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass)
{
	using std::uniform_real_distribution;
	uniform_real_distribution<float> randAngle (0.0, 200.0*PI);
	uniform_real_distribution<float> randRadius (INNER_BOUND, SYSTEM_SIZE);
	uniform_real_distribution<float> randHeight (0.0, SYSTEM_THICKNESS);
	std::default_random_engine gen (0);
	float angle;
	float radius;
	float velocity;

	//STARS
	velocity = 0.67*sqrt((G*SOLAR_MASS)/(4*BINARY_SEPARATION*TO_METERS));
	//STAR 1
	xpos[0] = 0.0;///-BINARY_SEPARATION;
	ypos[0] = 0.0;
	zpos[0] = 0.0;
	xvel[0] = 0.0;
	yvel[0] = 0.0;//velocity;
	zvel[0] = 0.0;
	mass[0] = SOLAR_MASS;

	    ///STARTS AT NUMBER OF STARS///
	float totalExtraMass = 0.0;
	for (int i=1; i<NUM_BODIES; i++)
	{
		angle = randAngle(gen);
		radius = sqrt(SYSTEM_SIZE)*sqrt(randRadius(gen));
		velocity = pow(((G*(SOLAR_MASS+((radius-INNER_BOUND)/SYSTEM_SIZE)*EXTRA_MASS*SOLAR_MASS))
					  	  	  	  	  / (radius*TO_METERS)), 0.5);
		xpos[i] =  radius*cos(angle);
		ypos[i] =  radius*sin(angle);
		zpos[i] =  randHeight(gen)-SYSTEM_THICKNESS/2;
		xvel[i] =  velocity*sin(angle);
		yvel[i] = -velocity*cos(angle);
		zvel[i] =  0.0;
		mass[i] = (EXTRA_MASS*SOLAR_MASS)/NUM_BODIES;
		totalExtraMass += (EXTRA_MASS*SOLAR_MASS)/NUM_BODIES;
	}
	std::cout << "\nTotal Disk Mass: " << totalExtraMass;
	std::cout << "\nEach Particle weight: " << (EXTRA_MASS*SOLAR_MASS)/NUM_BODIES
			  << "\n______________________________\n";
}

void runSimulation(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, char* image, float* hdImage)
{
	int nBlocks=(NUM_BODIES+1024-1)/1024;
	//createFirstFrame
	renderClear(image, hdImage);
	GPUrenderBodies<<<nBlocks+1,1024>>>(xpos,ypos,zpos,xvel,yvel,zvel,mass,hdImage);
	writeRender(image, hdImage, 1);
	
	for (int step=1; step<STEP_COUNT; step++)
	{
		std::cout << "\nBeginning timestep: " << step;
		printf("\nStartK\n");
		interactBodies<<<nBlocks,1024>>>(xpos,ypos,zpos,xvel,yvel,zvel,mass);
		hipDeviceSynchronize();
		printf("EndK\n");
		//printf("EndK2\n");

		if (step%RENDER_INTERVAL==0)
		{
			std::cout << "\nWriting frame " << step;
			if (DEBUG_INFO)	{std::cout << "\nClearing Pixels..." << std::flush;}
			renderClear(image, hdImage);
			if (DEBUG_INFO) {std::cout << "\nRendering Particles..." << std::flush;}
			//renderBodies(pos, vel, hdImage);
			GPUrenderBodies<<<nBlocks+1,1024>>>(xpos,ypos,zpos,xvel,yvel,zvel,mass,hdImage);
			hipDeviceSynchronize();
			if (DEBUG_INFO) {std::cout << "\nWriting frame to file..." << std::flush;}
			writeRender(image, hdImage, step);
		}
		if (DEBUG_INFO) {std::cout << "\n-------Done------- timestep: "
			       << step << "\n" << std::flush;}
	}
}
__global__ void interactBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < NUM_BODIES)
	{		
		float Fx=0.0f; float Fy=0.0f; float Fz=0.0f;
		float xposi=xpos[i];
		float yposi=ypos[i];
		float zposi=zpos[i];
		#pragma unroll
		for(int j=0; j < NUM_BODIES; j++)
		{
			if(i!=j)
			{ 
				vec3 posDiff;
				posDiff.x = (xposi-xpos[j])*TO_METERS;
				posDiff.y = (yposi-ypos[j])*TO_METERS;
				posDiff.z = (zposi-zpos[j])*TO_METERS;
				float dist = sqrt(posDiff.x*posDiff.x+posDiff.y*posDiff.y+posDiff.z*posDiff.z);
				float F = TIME_STEP*(G*mass[i]*mass[j]) / ((dist*dist + SOFTENING*SOFTENING) * dist);
				//float Fa = F/mass[i];
				Fx-=F*posDiff.x;
				Fy-=F*posDiff.y;
				Fz-=F*posDiff.z;
			}	
		}
		xvel[i] += Fx/mass[i];
		yvel[i] += Fy/mass[i];
		zvel[i] += Fz/mass[i];
		xpos[i] += TIME_STEP*xvel[i]/TO_METERS;
		ypos[i] += TIME_STEP*yvel[i]/TO_METERS;
		zpos[i] += TIME_STEP*zvel[i]/TO_METERS;
	}
}

float magnitude(vec3 v)
{
	return sqrt(v.x*v.x+v.y*v.y+v.z*v.z);
}

void renderClear(char* image, float* hdImage)
{
	for (int i=0; i<WIDTH*HEIGHT*3; i++)
	{
		image[i] = 0; //char(image[i]/1.2);
		hdImage[i] = 0.0;
	}
}

__global__ void GPUrenderBodies(float* xpos, float* ypos, float* zpos, float* xvel, float* yvel, float* zvel, float* mass, float* hdImage)
{
	/// ORTHOGONAL PROJECTION
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float velocityMax = MAX_VEL_COLOR; //35000
	float velocityMin = sqrt(0.8*(G*(SOLAR_MASS+EXTRA_MASS*SOLAR_MASS))/(SYSTEM_SIZE*TO_METERS)); //MIN_VEL_COLOR;
	if(i<NUM_BODIES)
	{
		float vxsqr=xvel[i]*xvel[i];
		float vysqr=yvel[i]*yvel[i];
		float vzsqr=zvel[i]*zvel[i];
		float vMag = sqrt(vxsqr+vysqr+vzsqr);
		int x = (WIDTH/2.0)*(1.0+xpos[i]/(SYSTEM_SIZE*RENDER_SCALE));
		int y = (HEIGHT/2.0)*(1.0+ypos[i]/(SYSTEM_SIZE*RENDER_SCALE));

		if (x>DOT_SIZE && x<WIDTH-DOT_SIZE && y>DOT_SIZE && y<HEIGHT-DOT_SIZE)
		{
			float vPortion = sqrt((vMag-velocityMin) / velocityMax);
			float xPixel = (WIDTH/2.0)*(1.0+xpos[i]/(SYSTEM_SIZE*RENDER_SCALE));
			float yPixel = (HEIGHT/2.0)*(1.0+ypos[i]/(SYSTEM_SIZE*RENDER_SCALE));
			float xP = floor(xPixel);
			float yP = floor(yPixel);
			color c;
			c.r = max(min(4*(vPortion-0.333),1.0),0.0);
                        c.g = max(min(min(4*vPortion,4.0*(1.0-vPortion)),1.0),0.0);
                        c.b = max(min(4*(0.5-vPortion),1.0),0.0);
			for (int a=-DOT_SIZE/2; a<DOT_SIZE/2; a++)
			{
				for (int b=-DOT_SIZE/2; b<DOT_SIZE/2; b++)
				{
					float cFactor = PARTICLE_BRIGHTNESS /(pow(exp(pow(PARTICLE_SHARPNESS*(xP+a-xPixel),2.0)) + exp(pow(PARTICLE_SHARPNESS*(yP+b-yPixel),2.0)),/*1.25*/0.75)+1.0);
					int pix = 3*(xP+a+WIDTH*(yP+b));
					hdImage[pix+0] += c.r*cFactor;
					hdImage[pix+1] += c.g*cFactor;
					hdImage[pix+2] += c.b*cFactor;
				}
			}
		}
	}
}

float clamp(float x)
{
	return max(min(x,1.0),0.0);
}

void writeRender(char* data, float* hdImage, int step)
{
	
	for (int i=0; i<WIDTH*HEIGHT*3; i++)
	{
		data[i] = int(255.0*clamp(hdImage[i]));
	}

	int frame = step/RENDER_INTERVAL + 1;//RENDER_INTERVAL;
	std::string name = "images/Step"; 
	int i = 0;
	if (frame == 1000) i++; // Evil hack to avoid extra 0 at 1000
	for (i; i<4-floor(log(frame)/log(10)); i++)
	{
		name.append("0");
	}
	name.append(std::to_string(frame));
	name.append(".ppm");

	std::ofstream file (name, std::ofstream::binary);

	if (file.is_open())
	{
//		size = file.tellg();
		file << "P6\n" << WIDTH << " " << HEIGHT << "\n" << "255\n";
		file.write(data, WIDTH*HEIGHT*3);
		file.close();
	}

}

